
#include <iostream>
#include <hip/hip_runtime.h>
#include <time.h>
#include <math.h>
using namespace std;

    // 테스트 용이므로 일단 자료 크기는 1000으로
    // 1D이니까 그냥 블럭사이즈는 512로
    // EP는 슬라이스의 사이즈

//10만개부터 에러났음. 아마 랜덤 숫자 만들어내는 데, 아니면 GPU메모리 상에서 문제가 발생한 것 같음.
// 만일 화면 데이터를 정렬한다고 하면, 2560x1600 = 4,096,000 픽셀이니까 GPU메모리 상에서의 문제가
// 아니라 랜덤 숫자 만들어내는 곳에서 문제가 발생한 것일 수도...
#define DATASIZE    100000
#define BLOCK_SIZE    512

__global__ void oddevensort ( int * input, unsigned int len, int i )
{
     //개별 블럭의 좌표
    unsigned int tx = threadIdx.x;

    //전체 이미지의 좌표
    unsigned int x = tx + blockDim.x * blockIdx.x;
    //이동에 쓸 임시 변수
    int temp;

    //자료의 길이만큼 돌리는데, 인덱스(i)가 짝수이면 데이터의 짝수자리와 그 다음 숫자를 비교.
    //인덱스가 홀수이면 데이터의 홀수자리와 그 다음 숫자를 비교해서 정렬한다.
    if( i % 2 == 0 )
    {
    		// 길이를 측정안해주면 블럭에 남아있던 자리에 있는 자료가 튀어나올 수 있으니 조심.
    	if( input[x] > input[x+1] && x < len && x % 2 == 0)
    	{
    		temp = input[x+1];
    		input[x+1] = input[x];
        	input[x] = temp;
        }
    }
    else
    {
    	if( input[x] > input[x+1] && x < len && x % 2 != 0)
        {
    		temp = input[x+1];
        	input[x+1] = input[x];
        	input[x] = temp;
        }
    }
    	__syncthreads();
}



int main()
{
    // 테스트에 쓸 숫자 생성
    int TestInput[DATASIZE], TestOutput[DATASIZE];

    srand(time(NULL));

    for( int i = 0; i < DATASIZE; i++ )
    {
        TestInput[i] = rand() % 500;
    }

    //device 설정
    int *devInput, *devOutput;
    //일단 크기는 아니까
    unsigned int MemDataSize = DATASIZE * sizeof(int);

    // device 자리 잡아주고
    hipMalloc((void**)&devInput, MemDataSize );
    hipMalloc((void**)&devOutput, MemDataSize );
    hipMemset( devOutput, 0, MemDataSize );

    // 자리 잡았으면 복사
    hipMemcpy( devInput, TestInput, MemDataSize, hipMemcpyHostToDevice);

    // block 크기 설정
    // 1D 이니까, 그냥 간단하게...
    dim3    dimBlocksize( BLOCK_SIZE );
    dim3    dimGridsize( ceil((DATASIZE-1)/(float)BLOCK_SIZE) + 1 );
    // 일단 Max값과 min값을 알아내야됨.
    // 처음부터 끝까지 휙 둘러보면 되니 이건 CPU에게 맡김.

    for( int i=0; i<DATASIZE; i++)
    {
    	oddevensort<<< dimGridsize, dimBlocksize >>>( devInput, DATASIZE, i );
    }

    // 결과물 복사
    hipMemcpy( TestOutput, devInput, MemDataSize, hipMemcpyDeviceToHost);

    for( int i=0; i<DATASIZE; i++ )
    {
    	cout << TestOutput[i] << ", ";
    	if( (i+1)%10 == 0 )
    	{
    		cout << endl;
    	}
    }

    // 위에 GPU에 마련한 자리 해소. 그때 그때 해놓는 게 편할 듯
    hipFree( devInput );
    hipFree( devOutput );

    return 0;
}
